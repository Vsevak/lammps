#include "hip/hip_runtime.h"
/*
 * lal_sort.cu
 *
 *  Created on: Aug 14, 2020
 *      Author: vsevak
 */

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#endif

#ifdef __CDT_PARSER__
#include "lal_precision.h"
#include "lal_preprocessor.h"
#include "lal_aux_fun1.h"
#endif
// This value must be consistent with lal_sort.h
#define BLOCK 256

__kernel void k_local(
    __global unsigned *k,
    __global int *v,
    __global unsigned *k_out,
    __global unsigned *v_out,
    const int n,
    __global unsigned *prefix,
    __global unsigned *block,
    const int b) {

  __local unsigned int l_input[BLOCK];
  __local          int l_input_value[BLOCK];
  __local unsigned int l_mask[BLOCK+1];
  __local unsigned int l_merged[BLOCK];
  __local unsigned int l_mask_sums[4];
  __local unsigned int l_scan_mask_sums[4];

  int tid = THREAD_ID_X;
  int gid = GLOBAL_ID_X;
  // Read key from global to shared
  if (gid < n) {
    l_input[tid] = k[gid];
  } else {
    l_input[tid] = 0;
  }
  __syncthreads();
  unsigned int input_key = l_input[tid];
  // Read value from global to register
  int input_value;
  if (gid < n) {
    input_value = v[gid];
  } else {
    input_value = 0;
  }
  // Get two LSB
  unsigned int get_two_bits = (input_key >> b ) & 3;

  // 4-way (2-bit) radix sort
  for(unsigned int i = 0; i < 4; ++i) {
    l_mask[tid] = 0;
    if (tid == 0) {
      l_mask[BLOCK] = 0;
    }
    __syncthreads();

    bool out_eq_in = false;
    if (gid < n) {
      out_eq_in = (get_two_bits == i);
      l_mask[tid] = out_eq_in;
    }
    __syncthreads();

    // Prefix-sum masks
    // Hillis and Steele since it is just within shared memory and
    // the number of threads is equal to the number of elements
    int target = 0;
    unsigned int sum = 0;
    unsigned int total = (unsigned int) log2((float)BLOCK);
    for (unsigned int step = 0; step < total; ++step) {
      target = tid - (1 << step);
      if (target >= 0) {
        sum = l_mask[tid] + l_mask[target];
      } else {
        sum = l_mask[tid];
      }
      __syncthreads();
      l_mask[tid] = sum;
      __syncthreads();
    }

    // Shift
    unsigned int buffer = l_mask[tid];
    __syncthreads();
    l_mask[tid + 1] = buffer;
    __syncthreads();
    if (tid == 0) {
      l_mask[0] = 0;
      unsigned int tsum = l_mask[BLOCK];
      l_mask_sums[i] = tsum;
      // gridDim.x expressed in terms of the Geryon
      int grid = GLOBAL_SIZE_X;
      grid /= BLOCK_SIZE_X;
      block[i * grid  + BLOCK_ID_X]  = tsum;
    }
    __syncthreads();

    if (out_eq_in && (gid < n)) {
      l_merged[tid] = l_mask[tid];
    }

    __syncthreads();
  }

  if (tid == 0) {
    unsigned int csum = 0;
    // Serial scan the resulting masks
    for (int i = 0; i < 4; ++ i){
      l_scan_mask_sums[i] = csum;
      csum += l_mask_sums[i];
    }
  }
  __syncthreads();

  if (gid < n) {
    unsigned int merged = l_merged[tid];
    int pos = merged + l_scan_mask_sums[get_two_bits];
    __syncthreads();
    l_input[pos] = input_key;
    l_input_value[pos] = input_value;
    l_merged[pos] = merged;
    __syncthreads();

    // Global output
    prefix[gid] = l_merged[tid];
    k_out[gid] = l_input[tid];
    v_out[gid] = l_input_value[tid];
  }
}

__kernel void k_global_scatter(
    __global unsigned int *key_out,
    __global int * value_out,
    __global unsigned int *key_in,
    __global int * value_in,
    const int n,
    __global unsigned int *prefix,
    __global unsigned int *scan_block,
    const int b) {

  int gid = GLOBAL_ID_X;
  // gridDim.x expressed in terms of Geryon
  int grid = GLOBAL_SIZE_X; grid /= BLOCK_SIZE_X;

  if (gid < n) {
    unsigned int k_in = key_in[gid];
    unsigned int get_two_bits = (k_in >> b) & 3;
    int v_in = value_in[gid];
    unsigned int pref = prefix[gid];
    unsigned int pos = scan_block[get_two_bits * grid + BLOCK_ID_X] + pref;
    __syncthreads();
    key_out[pos] = k_in;
    value_out[pos] = v_in;
  }
}

__kernel void k_check(
    __global int * value_in,
    const int n,
    __global int *results) {
  int gid = GLOBAL_ID_X;
  int tid = THREAD_ID_X;
  __local int val[BLOCK+1];
  __local int flag[BLOCK];
  if (gid < n) {
    val[tid] = value_in[gid];
  }
  if (tid == 0 && (gid + BLOCK) < n) {
    val[BLOCK] = value_in[gid + BLOCK];
  }
  __syncthreads();
  if(gid < n) {
    flag[tid] = val[tid] > val[tid+1];
  } else {
    flag[tid] = 0;
  }
  __syncthreads();

  for (unsigned int s = BLOCK/2; s > 0; s>>=1) {
    if (tid < s) {
      flag[tid] += flag[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0) {
    results[BLOCK_ID_X] = flag[0];
  }
}

