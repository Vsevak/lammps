#include "hip/hip_runtime.h"
/*
 * lal_sort.cu
 *
 *  Created on: Aug 14, 2020
 *      Author: vsevak
 */

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#endif

#ifdef __CDT_PARSER__
#include "lal_precision.h"
#include "lal_preprocessor.h"
#include "lal_aux_fun1.h"
#endif

#define BLOCK 256

__kernel void k_local(__global unsigned *restrict k,
    __global int *restrict v,
    __global unsigned *restrict out,
    const int n,
    __global unsigned *restrict prefix,
    __global unsigned *restrict block,
    const int b) {

  __local unsigned int l_input[BLOCK];
  __local unsigned int l_mask[BLOCK+1];
  __local unsigned int l_merged[BLOCK];
  __local unsigned int l_mask_sums[4];
  __local unsigned int l_scan_mask_sums[4];

  int tid = THREAD_ID_X;
  int gid = GLOBAL_ID_X;
  // Read key from global to shared
  if (gid < n) {
    l_input[tid] = k[gid];
  } else {
    l_input[tid] = 0;
  }
  __syncthreads();
  unsigned int thread_input = l_input[tid];
  // Get two LSB
  unsigned int get_two_bits = (thread_input >> b ) & 3;

  // 4-way radix
  for(int i=0; i<4; ++i) {
    l_mask[tid] = 0;
    if (tid == 0) {
      l_mask[BLOCK] = 0;
    }
    __syncthreads();

    bool out_eq_in = false;
    if (gid < n) {
      out_eq_in = (get_two_bits == i);
      l_mask[tid] = out_eq_in;
    }
    __syncthreads();

    int target = 0;
    unsigned int sum = 0;
    int total = (int) log2f(BLOCK);
    // Prefix-sum masks
    for (int step = 0; step < total; ++step) {
      target = tid - (1 << step);
      sum = l_mask[tid] + ((target >= 0) ? l_mask[target] : 0);
      __syncthreads();
      l_mask[tid] = sum;
      __syncthreads();
    }

    // Shift
    unsigned int buffer = l_mask[tid];
    __syncthreads();
    l_mask[tid + 1] = buffer;
    __syncthreads();
    if (tid ==0) {
      l_mask[0] = 0;
      sum = l_mask[BLOCK];
      l_mask_sums[i] = sum;
      int grid = GLOBAL_SIZE_X;
      grid /= BLOCK_SIZE_X;
      block[i * grid  + BLOCK_ID_X]  = sum;
    }
    __syncthreads();

    if (out_eq_in && (gid < n)) {
      l_merged[tid] = l_mask[tid];
    }
    __syncthreads();
  }

  // Serial scan resulting masks
  if (tid == 0) {
    unsigned int csum = 0;
    for (int i = 0; i < 4; ++ i){
      l_scan_mask_sums[i] = csum;
      csum += l_mask_sums[i];
    }
  }
  __syncthreads();

  if (gid < n) {
    unsigned int merged = l_merged[tid];
    unsigned int pos = merged + l_scan_mask_sums[get_two_bits];
    __syncthreads();
    l_input[pos] = thread_input;
    l_merged[pos] = merged;
    __syncthreads();

    // Global output
    prefix[gid] = l_merged[tid];
    out[gid] = l_input[tid];
  }
}

__kernel void k_global(__global int *restrict x, const int n){
  int tid = BLOCK_ID_X*BLOCK_SIZE_X + THREAD_ID_X;
  if (tid < n){
    //x[tid] *= 100;
  }
}

