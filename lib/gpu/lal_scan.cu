#include "hip/hip_runtime.h"
/*
 * lal_scan.cu
 *
 *  Created on: Aug 17, 2020
 *      Author: vsevak
 */

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#endif

#ifdef __CDT_PARSER__
#include "lal_precision.h"
#include "lal_preprocessor.h"
#include "lal_aux_fun1.h"
#endif
// This value must be consistent with lal_scan.h
// BLOCK = Scan::block_size
#define BLOCK 256

__kernel void k_scan (
    __global unsigned int *input,
    __global unsigned int *output,
    const int n,
    __global unsigned int *block) {

  __local unsigned int l_tmp[BLOCK+1];
  int tid = THREAD_ID_X;
  int gid = GLOBAL_ID_X;

  if (gid < n) {
    l_tmp[tid] = input[gid];
  } else {
    l_tmp[tid] = 0;
  }
  __syncthreads();
  int target = 0;
  unsigned int sum = 0;
  int total = (int) log2((float)BLOCK);
  for (unsigned int step = 0; step < total; ++step) {
    target = tid - (1 << step);
    if (target >= 0) {
      sum = l_tmp[tid] + l_tmp[target];
    } else {
      sum = l_tmp[tid];
    }
    __syncthreads();
    l_tmp[tid] = sum;
    __syncthreads();
  }


  // Shift
  if (tid == 0) {
    sum = 0;
    block[BLOCK_ID_X] = l_tmp[BLOCK-1];
    //      l_mask_sums[i] = tsum;
    //      // gridDim.x expressed in terms of the Geryon
  } else {
    sum = l_tmp[tid - 1];
  }
  if (gid < n) {
    output[gid] = sum;
  }
}

__kernel void k_add (
    __global unsigned int *plus,
    __global unsigned int *result,
    const int n) {

  int tid = THREAD_ID_X;
  int gid = GLOBAL_ID_X;
  int block = BLOCK_ID_X;
  __local unsigned int block_inc;
  if (tid == 0) {
    block_inc = plus[block];
  }
  __syncthreads();
  if (gid < n) {
    result[gid] += block_inc;
  }
}

